#include "hip/hip_runtime.h"
#include "utility.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "numgen.c"

__host__
void errorexit(const char *s) {
    printf("\n%s",s);	
    exit(EXIT_FAILURE);	 	
}

__device__
bool is_prime(int n) {
    if (n <= 1) return false;
    if (n == 2) return true;
    if (n % 2 == 0) return false;
    for (int i = 3; i <= sqrtf((float)n); i += 2) {
        if (n % i == 0) return false;
    }
    return true;
}

__global__ 
void check_primes(int *results, int size) {
    int my_index = blockIdx.x * blockDim.x + threadIdx.x;
    if (my_index < size) {
        results[my_index] = is_prime(my_index);
    }
}

int isPrime(long unsigned int n) {
  if (n == 0 || n == 1) {
    return false;
  }
  if (n == 2) {
    return true;
  }
  for (unsigned long int i = 2; i * i <= n;i++) {
    if (n % i == 0) {
      return false;
    }
  }
  return true;
}

int main(int argc,char **argv) {

  Args ins__args;
  parseArgs(&ins__args, &argc, argv);
  
  //program input argument
  long inputArgument = ins__args.arg; 
  unsigned long int *numbers = (unsigned long int*)malloc(inputArgument * sizeof(unsigned long int));
  numgen(inputArgument, numbers);

  struct timeval ins__tstart, ins__tstop;
  gettimeofday(&ins__tstart, NULL);
  
  int threadsinblock=1024;
  int blocksingrid = (inputArgument + threadsinblock - 1) / threadsinblock;	

  int size = threadsinblock*blocksingrid;
  int *hresults=(int*)malloc(size*sizeof(int));
  if (!hresults) errorexit("Error allocating memory on the host");	

  int *dresults=NULL;
  if (hipSuccess!=hipMalloc((void **)&dresults,size*sizeof(int)))
    errorexit("Error allocating memory on the GPU");

  check_primes<<<blocksingrid, threadsinblock>>>(dresults, inputArgument);
  if (hipSuccess != hipGetLastError())
      errorexit("Error during kernel launch");
  
  if (hipSuccess!=hipMemcpy(hresults,dresults,size*sizeof(int),hipMemcpyDeviceToHost))
    errorexit("Error copying results");
  
  int prime_count = 0;
  for(int i = 0; i <= inputArgument; i++) {
    if (hresults[i]) {
      prime_count++;
    }
  }

  printf("\nFound %d primes in total\n", prime_count);

  free(hresults);
  if (hipSuccess!=hipFree(dresults))
    errorexit("Error when deallocating space on the GPU");

  gettimeofday(&ins__tstop, NULL);
  ins__printtime(&ins__tstart, &ins__tstop, ins__args.marker);

}
